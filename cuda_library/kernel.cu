#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "cuda_context.hpp"
#include "cuda_pointer.hpp"
#include "cuda_pointer_vector_utils.hpp"
#include "cuda_multi_dim_vector.hpp"
#include "cuda_pointer_thrust_utils.hpp"
#include "cuda_multi_dim_vector_thrust_utils.hpp"
#include "multi_dim_vector_with_memory.hpp"

#include "compile_time_differentiation_tests.hpp"

#include <thrust/functional.h>
#include <thrust/sort.h>
#include <assert.h>

#include <memory>
namespace aks
{
    template<typename _value_type, size_t _dimensions>
    struct cuda_multi_dim_vector
    {
        typedef _value_type value_type;
        enum{ dimensions = _dimensions };
        typedef multi_dim_vector<value_type, dimensions> device_data_type;


    private:
        cuda_pointer<_value_type> m_data;
    };
}

hipError_t addWithCuda(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b);

__global__ void addKernel(aks::multi_dim_vector<int, 1> c, aks::multi_dim_vector<int const, 1> a, aks::multi_dim_vector<int const, 1> b)
{
    int i = threadIdx.x;
    c(i) = a(i) + b(i);
}

template<typename T, size_t N>
using host_multi_dim_vector = aks::multi_dim_vector_with_memory<T, N, std::vector<T>>;

template<typename T, size_t N>
using cuda_multi_dim_vector = aks::multi_dim_vector_with_memory<T, N, aks::cuda_pointer<T>>;

void check2()
{
	compile_time_differentiation_tests();
	{
		host_multi_dim_vector<int, 3> vec(3, 4, 5);
		auto view = vec.view();
		auto const& const_vec = vec;
		auto const_view = const_vec.view();
		printf("");
	}
	{  
		host_multi_dim_vector<int, 3> host_vec(3, 4, 5);
		auto host_view = host_vec.view();
		auto m0 = host_view.max_dimension<0>();
		auto m1 = host_view.max_dimension<1>();
		auto m2 = host_view.max_dimension<2>();
		auto m00 = aks::get_max_dim<0>(host_view);
		auto m01 = aks::get_max_dim<1>(host_view);
		auto m02 = aks::get_max_dim<2>(host_view);
		for(size_t x=0; x<3; ++x)
			for (size_t y = 0; y<4; ++y)
				for (size_t z = 0; z < 5; ++z)
				{
					host_view(x, y, z) = x*4*5 + y*5 + z;
				}

		cuda_multi_dim_vector<int, 3> vec(host_vec.view().data(), 3, 4, 5);
		auto view = vec.view();
		auto const& const_vec = vec;
		auto const_view = const_vec.view();

		std::vector<int> ret(view.total_size());
		vec.m_data.load(ret.data());

		printf("");
	}
}

int main()
{
	check2();
    //aks::cuda_context ctxt;
    
    std::vector<int> const a = { 1, 2, 3, 4, 5 };
    std::vector<int> const b = { 10, 20, 30, 40, 50 };
    std::vector<int> c;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda2(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b)
{
    using namespace aks;

    cuda_context ctxt(cuda_device(0));
    auto da = make_cuda_pointer(a);
    auto db = make_cuda_pointer(b);
    //cuda_pointer<int const> db(size, b);
    cuda_pointer<int> dc(a.size());
    //dc.deep_copy_from(db);
    auto const ma = make_multi_dim_vector(da.data(), da.size());
    auto const mb = make_multi_dim_vector(db.data(), da.size());
    auto mc = make_multi_dim_vector(dc.data(), da.size());

    {
        cuda_sync_context sync_ctxt;
        addKernel<<<1, a.size()>>>(mc, ma, mb);
    }

    thrust::transform(thrust_utils::begin(ma), thrust_utils::end(ma), thrust_utils::begin(mb), thrust_utils::begin(mc), thrust::plus<int>());

    printf("%d\n", thrust::reduce(thrust_utils::begin(ma), thrust_utils::end(ma), (int)0));
    printf("%d\n", thrust::reduce(thrust_utils::begin(mb), thrust_utils::end(mb), (int)0));
    printf("%d\n", thrust::reduce(thrust_utils::begin(mc), thrust_utils::end(mc), (int)0));

    thrust::transform(thrust_utils::begin(mc), thrust_utils::end(mc), thrust_utils::begin(mc), thrust::negate<int>());
    thrust::sort(thrust_utils::begin(mc), thrust_utils::end(mc));	

    c = from_cuda_pointer(dc);

    assert(!da.has_error_occurred() && !db.has_error_occurred() && !dc.has_error_occurred());

    //check();

    return last_status();

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b)
{
    aks::cuda_context ctxt(aks::cuda_device(0));
    aks::cuda_pointer<int const> da = aks::make_cuda_pointer(a);
    aks::cuda_pointer<int const> db = aks::make_cuda_pointer(b);      
    //aks::cuda_pointer<int const> db(size, b);
    aks::cuda_pointer<int> dc(a.size());
    //dc.deep_copy_from(db);
    aks::multi_dim_vector<int const, 1> const ma = aks::make_multi_dim_vector(da.data(), da.size());
    aks::multi_dim_vector<int const, 1> const mb = aks::make_multi_dim_vector(db.data(), da.size());
    aks::multi_dim_vector<int, 1> mc = aks::make_multi_dim_vector(dc.data(), da.size());

    {
        aks::cuda_sync_context sync_ctxt;
        addKernel <<<1, a.size()>>>(mc, ma, mb);
    }

    //thrust::device_vector<int> const tva = aks::to_thrust_device_vector(ma);
    //thrust::device_vector<int> const tvb = aks::to_thrust_device_vector(mb);
    //thrust::device_vector<int> tvc = aks::to_thrust_device_vector(mc);  

    thrust::transform(aks::thrust_utils::begin(ma), aks::thrust_utils::end(ma), aks::thrust_utils::begin(mb), aks::thrust_utils::begin(mc), thrust::plus<int>());

    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(ma), aks::thrust_utils::end(ma), (int)0));
    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(mb), aks::thrust_utils::end(mb), (int)0));
    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc), (int)0));

    thrust::transform(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc), aks::thrust_utils::begin(mc), thrust::negate<int>());
    thrust::sort(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc));

    c = aks::from_cuda_pointer(dc);

    assert(!da.has_error_occurred() && !db.has_error_occurred() && !dc.has_error_occurred());

    //check();

    return aks::last_status();
    
}
