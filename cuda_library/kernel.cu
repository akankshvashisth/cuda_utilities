#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "cuda_context.hpp"
#include "cuda_pointer.hpp"
#include "cuda_pointer_vector_utils.hpp"
#include "cuda_multi_dim_vector.hpp"
#include "cuda_pointer_thrust_utils.hpp"
#include "cuda_multi_dim_vector_thrust_utils.hpp"
#include "multi_dim_vector_with_memory.hpp"

#include "compile_time_differentiation_tests.hpp"
#include "experiments.hpp"

#include <thrust/functional.h>
#include <thrust/sort.h>
#include <assert.h>

#include <memory>
#include "multi_dim_vector.hpp"
#include "multi_dim_vector_iterator.hpp"
#include "multi_dim_vector_range.hpp"
#include "cuda_blas_manager.hpp"
#include "cuda_blas_level_1.hpp"
#include "cuda_blas_level_2.hpp"

hipError_t addWithCuda(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b);
hipError_t addWithCuda2(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b);

__global__ void addKernel(aks::multi_dim_vector<int, 1> c, aks::multi_dim_vector<int const, 1> a, aks::multi_dim_vector<int const, 1> b)
{
    int i = threadIdx.x;
	int sum = 0;
	for (auto it = aks::begin(a, aks::token(5)), end = aks::end(a, aks::token(5)); it != end; ++it)
		sum += *it;
	for (auto const& x : aks::make_multi_dim_vector_range(b, aks::token()))
		sum += x;
    c(i) = a(i) + b(i) - sum;
}

__global__ void addKernel(aks::multi_dim_vector<int, 3> c, aks::multi_dim_vector<int const, 3> const a, aks::multi_dim_vector<int const, 3> const b)
{
	int const i = threadIdx.x;
	int const j = threadIdx.y;
	int const k = threadIdx.z;

	int sum = 0;
	for (auto it = aks::begin(a, aks::token(), j, k), end = aks::end(a, aks::token(), j, k); it != end; ++it)
		sum += *it;
	for (auto it = aks::begin(a, i, aks::token(), k), end = aks::end(a, i, aks::token(), k); it != end; ++it)
		sum += *it;
	for (auto const& x : aks::make_multi_dim_vector_range(b, i, j, aks::token()))
		sum += x;

	c(i, j, k) = sum;
}

void check2()
{
	compile_time_differentiation_tests();
	{
		aks::host_multi_dim_vector<int, 3> vec(3, 4, 5);
		auto view = vec.view();
		auto const& const_vec = vec;
		auto const_view = const_vec.view();
		printf("");
	}
	if(true)
	{  
		aks::cuda_context ctxt(aks::cuda_device(0));
		aks::host_multi_dim_vector<int, 3> host_vec(3, 4, 5);
		auto host_view = host_vec.view();
		auto m0 = host_view.max_dimension<0>();
		auto m1 = host_view.max_dimension<1>();
		auto m2 = host_view.max_dimension<2>();
		auto m00 = aks::get_max_dim<0>(host_view);
		auto m01 = aks::get_max_dim<1>(host_view);
		auto m02 = aks::get_max_dim<2>(host_view);
		for(size_t x=0; x<3; ++x)
			for (size_t y = 0; y<4; ++y)
				for (size_t z = 0; z < 5; ++z)
				{
					host_view(x, y, z) = x*4*5 + y*5 + z;
				}

		aks::cuda_multi_dim_vector<int, 3> vec = aks::to_device(host_vec);// (host_vec.view().data(), 3, 4, 5);
		aks::cuda_multi_dim_vector<int, 3> res(3, 4, 5);
		
		dim3 threadsPerBlock(3, 4, 5);
		{
			aks::cuda_sync_context sync_ctxt;
			addKernel<<< 1, threadsPerBlock >>>(res.view(), vec.view(), vec.cview());
		}

		//auto view = vec.view();
		//auto const& const_vec = vec;
		//auto const_view = const_vec.view();

		//auto tmp = aks::from_cuda_pointer(vec.m_data);

		//std::vector<int> ret(view.total_size());
		//vec.m_data.load(ret.data());

		aks::host_multi_dim_vector<int, 3> ret_vec(3, 4, 5);
		ret_vec << res;

		auto ret_vec2 = aks::to_host(res);
		
		printf("");
	}
}


void blas_checks()
{
    
    using namespace aks;
    using namespace aks::cuda_blas;
    cuda_context ctxt(cuda_device(0));
    cuda_blas_manager blas_mgr;

    std::vector<double> const a = { 1., 2., 3., 4., 50., 8., -9., 23.0 };
    aks::cuda_multi_dim_vector<double, 1> cuda_vec(a.data(), std::tuple<size_t>(a.size()));    
    int value = abs_max_index(blas_mgr, cuda_vec);
    std::cout << a[value] << std::endl;
    value = abs_min_index(blas_mgr, cuda_vec);
    std::cout << a[value] << std::endl;
    double sum = abs_sum(blas_mgr, cuda_vec);
    std::cout << sum << std::endl;
    std::vector<double> const b = { 1., -2.,  3. };
    std::vector<double> const c = { 2.,  2.,  2. };
    aks::cuda_multi_dim_vector<double, 1> cuda_vecb(b.data(), std::tuple<size_t>(b.size()));
    aks::cuda_multi_dim_vector<double, 1> cuda_vecc(c.data(), std::tuple<size_t>(c.size()));
    double dt = dot(blas_mgr, cuda_vecb, cuda_vecc);
    std::cout << dt << std::endl;
    std::cout << norm_sq(blas_mgr, cuda_vecb) << std::endl;
    scale_in_place(blas_mgr, cuda_vecb, 1.5);
    auto const dev_vecb = to_host(cuda_vecb);
    for (auto it = dev_vecb.view().cbegin(), end = dev_vecb.view().cend(); it != end; ++it) {
        std::cout << *it << ",";
    }
    std::cout << std::endl;
    {
        host_multi_dim_vector<double, 2> Avec(2, 5);
        host_multi_dim_vector<double, 1> xvec(5);
        auto A = Avec.view();
        auto x = xvec.view();
        A(0, 0) = 1.0;
        A(0, 1) = 2.0;
        A(0, 2) = 3.0;
        A(0, 3) = 4.0;
        A(0, 4) = 5.0;
        A(1, 0) = 6.0;
        A(1, 1) = 7.0;
        A(1, 2) = 8.0;
        A(1, 3) = 9.0;
        A(1, 4) = 10.0;
        x(0) = 1.5;
        x(1) = 2.5;
        x(2) = 3.5;
        x(3) = 4.5;
        x(4) = 5.5;
        auto devA = to_device(Avec);
        auto devx = to_device(xvec);
        auto devy(matrix_multiply(blas_mgr, devA, devx));
        auto yvec = to_host(devy);
        for (auto it = yvec.view().cbegin(), end = yvec.view().cend(); it != end; ++it) {
            std::cout << *it << ",";
        }
        std::cout << std::endl;
    }
}

int main()
{
    blas_checks();
    return 0;
    //main2();
    compile_time_differentiation_tests();
	run_experiments();
	//check2();

    //aks::cuda_context ctxt;
    
    std::vector<int> const a = { 1, 2, 3, 4, 5 };
    std::vector<int> const b = { 10, 20, 30, 40, 50 };
    std::vector<int> c;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda2(c, a, b);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda2(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b)
{
    using namespace aks;

    cuda_context ctxt(cuda_device(0));
    auto da = make_cuda_pointer(a);
    auto db = make_cuda_pointer(b);
    //cuda_pointer<int const> db(size, b);
    cuda_pointer<int> dc(a.size());
    //dc.deep_copy_from(db);
    auto const ma = make_multi_dim_vector(da.data(), da.size());
    auto const mb = make_multi_dim_vector(db.data(), da.size());
    auto mc = make_multi_dim_vector(dc.data(), da.size());

    {
        cuda_sync_context sync_ctxt;
        addKernel<<<1, a.size()>>>(mc, ma, mb);
    }

    thrust::transform(thrust_utils::begin(ma), thrust_utils::end(ma), thrust_utils::begin(mb), thrust_utils::begin(mc), thrust::plus<int>());

    printf("%d\n", thrust::reduce(thrust_utils::begin(ma), thrust_utils::end(ma), (int)0));
    printf("%d\n", thrust::reduce(thrust_utils::begin(mb), thrust_utils::end(mb), (int)0));
    printf("%d\n", thrust::reduce(thrust_utils::begin(mc), thrust_utils::end(mc), (int)0));

    thrust::transform(thrust_utils::begin(mc), thrust_utils::end(mc), thrust_utils::begin(mc), thrust::negate<int>());
    thrust::sort(thrust_utils::begin(mc), thrust_utils::end(mc));	

    c = from_cuda_pointer(dc);

    assert(!da.has_error_occurred() && !db.has_error_occurred() && !dc.has_error_occurred());

    //check();

    return last_status();

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(std::vector<int>& c, std::vector<int> const& a, std::vector<int> const& b)
{
    aks::cuda_context ctxt(aks::cuda_device(0));
    aks::cuda_pointer<int const> da = aks::make_cuda_pointer(a);
    aks::cuda_pointer<int const> db = aks::make_cuda_pointer(b);      
    //aks::cuda_pointer<int const> db(size, b);
    aks::cuda_pointer<int> dc(a.size());
    //dc.deep_copy_from(db);
    aks::multi_dim_vector<int const, 1> const ma = aks::make_multi_dim_vector(da.data(), da.size());
    aks::multi_dim_vector<int const, 1> const mb = aks::make_multi_dim_vector(db.data(), da.size());
    aks::multi_dim_vector<int, 1> mc = aks::make_multi_dim_vector(dc.data(), da.size());

    {
        aks::cuda_sync_context sync_ctxt;
        addKernel <<<1, a.size()>>>(mc, ma, mb);
    }

	c = aks::from_cuda_pointer(dc);

    //thrust::device_vector<int> const tva = aks::to_thrust_device_vector(ma);
    //thrust::device_vector<int> const tvb = aks::to_thrust_device_vector(mb);
    //thrust::device_vector<int> tvc = aks::to_thrust_device_vector(mc);  

    thrust::transform(aks::thrust_utils::begin(ma), aks::thrust_utils::end(ma), aks::thrust_utils::begin(mb), aks::thrust_utils::begin(mc), thrust::plus<int>());

    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(ma), aks::thrust_utils::end(ma), (int)0));
    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(mb), aks::thrust_utils::end(mb), (int)0));
    printf("%d\n", thrust::reduce(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc), (int)0));

    thrust::transform(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc), aks::thrust_utils::begin(mc), thrust::negate<int>());
	thrust::sort(aks::thrust_utils::begin(mc), aks::thrust_utils::end(mc));

    c = aks::from_cuda_pointer(dc);

    assert(!da.has_error_occurred() && !db.has_error_occurred() && !dc.has_error_occurred());

    //check();

    return aks::last_status();
    
}
